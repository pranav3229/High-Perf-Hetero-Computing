/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   : 
 To build use  : nvcc deviceQuery.cu -o deviceQuery
 ============================================================================
 */
#include <stdio.h>
#include <hip/hip_runtime.h>

int main ()
{
  int deviceCount = 0;
  hipGetDeviceCount (&deviceCount);
  if (deviceCount == 0)
    printf ("No CUDA compatible GPU.\n");
  else
    {
      hipDeviceProp_t prop;
      for (int i = 0; i < deviceCount; i++)
        {
          hipGetDeviceProperties (&prop, i);
          printf ("Dev #%i is %s\n", i, prop.name);

        printf(" --- General Information for device %d ---\n", i);
	        printf("Name: %s\n", prop.name);
	        printf("Compute capability: %d.%d\n", prop.major, prop.minor);
	        printf("Clock rate: %d\n", prop.clockRate);
	        printf("Device copy overlap: ");
	        if (prop.deviceOverlap)
	            printf("Enabled\n");
	        else
	            printf("Disabled\n");
	        printf("Kernel execition timeout : ");
	        if (prop.kernelExecTimeoutEnabled)
	            printf("Enabled\n");
	        else
	            printf("Disabled\n");
	        printf(" --- Memory Information for device %d ---\n", i);
	        printf("Total global mem: %ld\n", prop.totalGlobalMem);
	        printf("Total constant Mem: %ld\n", prop.totalConstMem);
	        printf("Max mem pitch: %ld\n", prop.memPitch);
	        printf("Texture Alignment: %ld\n", prop.textureAlignment);
	        printf(" --- MP Information for device %d ---\n", i);
	        printf("Multiprocessor count: %d\n",
	               prop.multiProcessorCount);
	        printf("Shared mem per mp: %ld\n", prop.sharedMemPerBlock);
	        printf("Registers per mp: %d\n", prop.regsPerBlock);
	        printf("Threads in warp: %d\n", prop.warpSize);
	        printf("Max threads per block: %d\n",
              prop.maxThreadsPerBlock);
	        printf("Max thread dimensions: (%d, %d, %d)\n",
	               prop.maxThreadsDim[0], prop.maxThreadsDim[1],
	               prop.maxThreadsDim[2]);
	        printf("Max grid dimensions: (%d, %d, %d)\n",
	               prop.maxGridSize[0], prop.maxGridSize[1],
	               prop.maxGridSize[2]);



        }
    }
  return 1;
}
