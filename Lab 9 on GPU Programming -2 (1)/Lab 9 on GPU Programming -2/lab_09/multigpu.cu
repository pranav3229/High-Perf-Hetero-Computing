#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
__global__ void doSmt(int *data)
{
  // Simplification of above 
  int myID = ( blockIdx.z * gridDim.x * gridDim.y  + 
               blockIdx.y * gridDim.x + 
               blockIdx.x ) * blockDim.x + 
               threadIdx.x; 

  printf ("Hello world from %i\n", myID);
}

const int DATASIZE=1024;

int main ()
{

hipStream_t str[4];
  int *h_data[4], *d_data[4];
  

  int deviceCount = 0;
  hipGetDeviceCount (&deviceCount);
  if (deviceCount == 0)
    printf ("No CUDA compatible GPU.\n");
	  else
	    {     
	      for (int i = 0; i < deviceCount; i++)
	        {
				hipSetDevice(i);
				printf("Allocating memory on GPU %d\n", i);
	           hipStreamCreate(&(str[i]));
				h_data[i] = (int *)malloc(sizeof(int) * DATASIZE);
				hipMalloc((void ** )&(d_data[i]), sizeof(int) * DATASIZE);
	
				// inititalize h_data[i]....
	
				printf("Trasferring data to memory on GPU %d\n", i);
	
				hipMemcpyAsync(d_data[i], h_data[i], sizeof(int) * DATASIZE, hipMemcpyHostToDevice, str[i]);
			   
				doSmt <<< 10, 256, 0, str[i] >>> (d_data[i]);
				
				printf("Trasferring data to host memory from GPU %d\n", i);
				hipMemcpyAsync(h_data[i], d_data[i], sizeof(int) * DATASIZE, hipMemcpyDeviceToHost, str[i]);    
	        }
	    }
	  return 1;
	}
