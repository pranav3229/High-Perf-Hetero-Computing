#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Kernel function to perform rank sort on GPU
__global__ void rankSortKernel(int *input, int *output, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < N) {
        int rank = 0;
        for (int j = 0; j < N; ++j) {
            if (input[tid] > input[j]) {
                rank++;
            }
        }
        output[rank] = input[tid];
    }
}

// Function to perform rank sort on CPU
void rankSortCPU(int *input, int *output, int N) {
    for (int i = 0; i < N; ++i) {
        int rank = 0;
        for (int j = 0; j < N; ++j) {
            if (input[i] > input[j]) {
                rank++;
            }
        }
        output[rank] = input[i];
    }
}

// Function to shuffle an array using Fisher-Yates algorithm
void shuffleArray(int *array, int n) {
    srand(time(NULL));
    for (int i = n - 1; i > 0; --i) {
        int j = rand() % (i + 1);
        // Swap array[i] and array[j]
        int temp = array[i];
        array[i] = array[j];
        array[j] = temp;
    }
}

int main() {
    int N;
    printf("Enter size of array (N): ");
    scanf("%d", &N);

    // Generate unique random numbers
    int *h_A = (int *)malloc(N * sizeof(int));
    int *uniqueNumbers = (int *)malloc(N * sizeof(int));
    srand(time(NULL));
    int index = 0;

    while (index < N) {
        int randomNum = rand() % 100000; // Generate random numbers between 0 and 99
        int isUnique = 1;
        // Check if randomNum is unique
        for (int i = 0; i < index; ++i) {
            if (uniqueNumbers[i] == randomNum) {
                isUnique = 0;
                break;
            }
        }
        if (isUnique) {
            uniqueNumbers[index++] = randomNum;
        }
    }

    // Shuffle the unique numbers
    shuffleArray(uniqueNumbers, N);

    // Copy shuffled unique numbers to h_A
    for (int i = 0; i < N; ++i) {
        h_A[i] = uniqueNumbers[i];
    }

    // Print shuffled array
    printf("Randomized & Shuffled Array:\n");
    for (int i = 0; i < N; ++i) {
        printf("%d ", h_A[i]);
    }
    printf("\n");

    // Allocate memory on GPU
    int *d_A, *d_B;
    hipMalloc(&d_A, N * sizeof(int));
    hipMalloc(&d_B, N * sizeof(int));

    // Copy input array from host to device
    hipMemcpy(d_A, h_A, N * sizeof(int), hipMemcpyHostToDevice);

    // Determine block and grid dimensions
    int T, B;
    printf("Enter number of threads per block (T): ");
    scanf("%d", &T);
    printf("Enter number of blocks in grid (B): ");
    scanf("%d", &B);

    // Ensure that T * B is less than N
    if (T * B < N) {
        printf("Error: Number of threads * blocks must be >= number of elements in the array (N)\n");
        return 1;
    }

    // Launch GPU kernel and time GPU execution
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    rankSortKernel<<<B, T>>>(d_A, d_B, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float gpuTime = 0;
    hipEventElapsedTime(&gpuTime, start, stop);

    // Copy sorted array from device to host
    int *h_B = (int *)malloc(N * sizeof(int));
    hipMemcpy(h_B, d_B, N * sizeof(int), hipMemcpyDeviceToHost);

    // Perform rank sort on CPU and time CPU execution
    int *h_C = (int *)malloc(N * sizeof(int));
    clock_t cpuStart, cpuEnd;
    cpuStart = clock();
    rankSortCPU(h_A, h_C, N);
    cpuEnd = clock();
    float cpuTime = ((float)(cpuEnd - cpuStart)) / CLOCKS_PER_SEC * 1000;

    // Display results and timing
    printf("GPU Sorted Array:\n");
    for (int i = 0; i < N; ++i) {
        printf("%d ", h_B[i]);
    }
    printf("\n");
    printf("CPU Sorted Array:\n");
    for (int i = 0; i < N; ++i) {
        printf("%d ", h_C[i]);
    }
    printf("\n");

    printf("GPU Execution Time: %.8f ms\n", gpuTime);
    printf("CPU Execution Time: %.8f ms\n", cpuTime);

    // Compute and display speed-up factor
    float speedUp = cpuTime / gpuTime;
    printf("Speed-Up Factor (CPU vs GPU): %.8f\n", speedUp);

    // Clean up
    hipFree(d_A);
    hipFree(d_B);
    free(h_A);
    free(h_B);
    free(h_C);
    free(uniqueNumbers);

    return 0;
}
